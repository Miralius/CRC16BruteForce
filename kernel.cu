#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <iomanip>
#include "hip/hip_runtime.h"
#include ""

using namespace std::string_literals;

class cudaDeviceResetException : public std::exception
{
public:
    template <
        class StringType, 
        typename = std::enable_if_t
            <!std::is_base_of
                <cudaDeviceResetException,
                std::decay_t<StringType>
            >::value
        >
    >
    cudaDeviceResetException(StringType&& exceptionDescription)
        : std::exception(std::forward<StringType>(exceptionDescription).c_str())
    {}
};

class HEXByteSymbol
{
private:
    char firstSymbol;
    char secondSymbol;

public:
    HEXByteSymbol() = default;

    HEXByteSymbol(const char firstSymbol, const char secondSymbol) noexcept
        : firstSymbol(firstSymbol)
        , secondSymbol(secondSymbol)
    {}

    operator uint8_t()
    {
        auto stringHEXByte = ""s + firstSymbol + secondSymbol;
        return static_cast<uint8_t>(std::stoul(stringHEXByte, nullptr, 16));
    }
};

std::istream& operator>>(std::istream& in, HEXByteSymbol& obj)
{
    char firstSymbol{};
    char secondSymbol{};
    in >> firstSymbol >> secondSymbol;
    if (!in)
    {
        return in;
    }
    obj = HEXByteSymbol(firstSymbol, secondSymbol);
    return in;
}

class CRC16
{
private:
    uint16_t polynome{};
    uint16_t initValue{};
    uint16_t finalXORValue{};
    bool inputReflected{};
    bool resultReflected{};
    bool initialized{};
public:
    class device_type{};
    class host_type{};

    uint16_t getPolynome() const noexcept
    {
        return polynome;
    }
    uint16_t getInitValue() const noexcept
    {
        return initValue;
    }
    uint16_t getFinalXORValue() const noexcept
    {
        return finalXORValue;
    }
    bool isInputReflected() const noexcept
    {
        return inputReflected;
    }
    bool isResultReflected() const noexcept
    {
        return resultReflected;
    }
    bool isInitialized() const noexcept
    {
        return initialized;
    }

    CRC16() = default;

    __device__ CRC16(const uint16_t polynome, const uint16_t initValue, const uint16_t finalXORValue,
        const bool inputReflected, const bool resultReflected, device_type) noexcept
        : polynome(polynome)
        , initValue(initValue)
        , finalXORValue(finalXORValue)
        , inputReflected(inputReflected)
        , resultReflected(resultReflected)
        , initialized(true)
    {}

    CRC16(const uint16_t polynome, const uint16_t initValue, const uint16_t finalXORValue,
        const bool inputReflected, const bool resultReflected, host_type) noexcept
        : polynome(polynome)
        , initValue(initValue)
        , finalXORValue(finalXORValue)
        , inputReflected(inputReflected)
        , resultReflected(resultReflected)
        , initialized(true)
    {}
};

std::ostream& operator<<(std::ostream& out, const CRC16& data)
{
    return out << std::noshowbase << std::hex << std::uppercase
        << "Polynome: " << data.getPolynome() << ' '
        << "Init value: " << data.getInitValue() << ' '
        << "Final XOR value: " << data.getFinalXORValue() << ' ' << std::dec
        << "Input reflected: " << (data.isInputReflected() ? "yes" : "no") << ' '
        << "Result reflected: " << (data.isResultReflected() ? "yes" : "no");
}

bool yesOrNoToBool(const std::string& answer)
{
    if (answer == "yes")
    {
        return true;
    }
    else if (answer == "no")
    {
        return false;
    }
    else
    {
        throw std::invalid_argument("Wrong argument received!");
    }
}

std::istream& operator>>(std::istream& in, CRC16& obj)
{
    std::string description{};
    uint16_t polynome{};
    uint16_t initValue{};
    uint16_t finalXORValue{};
    std::string inputReflectedString{};
    std::string resultReflectedString{};
    in >> std::noshowbase >> std::hex >> std::uppercase >> description >> polynome >> description
        >> description >> initValue >> description >> description >> description >> finalXORValue
        >> description >> description >> inputReflectedString >> description >> description
        >> resultReflectedString;
    if (!in)
    {
        return in;
    }
    obj = CRC16(polynome, initValue, finalXORValue, yesOrNoToBool(inputReflectedString),
        yesOrNoToBool(resultReflectedString), CRC16::host_type());
    return in;
}

enum class inputResultReflected
{
    nothingReflected,
    inputReflected,
    resultReflected,
    inputAndResultReflected
};

template <typename T>
__device__ T reflect(const T& value)
{
    T reflectedByte{};
    auto bitCountOfType = sizeof(T) * 8;
    for (size_t i{}; i < bitCountOfType; i++)
    {
        uint8_t bitValue = value & (1 << i) ? 1 : 0;
        reflectedByte |= bitValue << ((bitCountOfType - 1) - i);
    }
    return reflectedByte;
}

__device__ uint16_t ComputeCRC16(const uint8_t* bytes, const size_t byteNumber, const uint16_t polynome,
    const uint16_t initValue, const uint16_t finalXorValue, const bool resultReflected)
{
    auto crc = initValue;
    for (size_t i{}; i < byteNumber; i++)
    {
        crc ^= (bytes[i] << 8);
        for (uint8_t j{}; j < 8; j++)
        {
            if ((crc & 0x8000) != 0)
            {
                crc = crc << 1 ^ polynome;
            }
            else
            {
                crc <<= 1;
            }
        }
    }
    if (resultReflected)
    {
        crc = reflect<uint16_t>(crc);
    }
    return (crc ^ finalXorValue);
}

__global__ void findCRC16Parameters(const uint8_t* data1, const uint8_t* data2, const uint8_t* data3,
    const uint8_t* data4, const uint8_t* reflectedData1, const uint8_t* reflectedData2, const uint8_t* reflectedData3,
    const uint8_t* reflectedData4, const uint16_t* crcs, const size_t size1, const size_t size2, const size_t size3,
    const size_t size4, const uint16_t finalXORValue, CRC16* result)
{
    uint16_t polynome = blockIdx.x * blockDim.x + threadIdx.x;
    uint16_t initValue = blockIdx.y * blockDim.y + threadIdx.y;
    auto inputResultReflectedType = static_cast<inputResultReflected>(blockIdx.z * blockDim.z + threadIdx.z);
    bool inputReflected{};
    auto resultReflected = inputResultReflectedType == inputResultReflected::resultReflected
        || inputResultReflectedType == inputResultReflected::inputAndResultReflected;
    if (inputResultReflectedType == inputResultReflected::inputReflected
        || inputResultReflectedType == inputResultReflected::inputAndResultReflected)
    {
        inputReflected = true;
    }
    if (ComputeCRC16(inputReflected ? reflectedData1 : data1, size1, polynome, initValue, finalXORValue,
        resultReflected) == crcs[0]
        && ComputeCRC16(inputReflected ? reflectedData2 : data2, size2, polynome, initValue, finalXORValue,
            resultReflected) == crcs[1]
        && ComputeCRC16(inputReflected ? reflectedData3 : data3, size3, polynome, initValue, finalXORValue,
            resultReflected) == crcs[2]
        && ComputeCRC16(inputReflected ? reflectedData4 : data4, size4, polynome, initValue, finalXORValue,
            resultReflected) == crcs[3])
    {
        *result = CRC16(polynome, initValue, finalXORValue, inputReflected, resultReflected, CRC16::device_type());
    }
}

template <class T, class StringType>
hipError_t cudaMallocAndMemcpyData(T*& pointer, const T& data, StringType&& string)
{
    hipError_t cudaStatus{};
    cudaStatus = hipMalloc(&pointer, sizeof(T));
    if (cudaStatus != hipSuccess)
    {
        throw std::runtime_error("hipMalloc for "s + string + " failed!\n" + hipGetErrorString(cudaStatus));
    }
    return hipMemcpy(pointer, &data, sizeof(T), hipMemcpyHostToDevice);
}

template <class T, class StringType>
hipError_t cudaMallocAndMemcpyData(T*& pointer, const std::vector<T>& data, StringType&& string)
{
    hipError_t cudaStatus{};
    cudaStatus = hipMalloc(&pointer, data.size() * sizeof(T));
    if (cudaStatus != hipSuccess)
    {
        throw std::runtime_error("hipMalloc for "s + string + " failed!\n" + hipGetErrorString(cudaStatus));
    }
    return hipMemcpy(pointer, data.data(), data.size() * sizeof(T), hipMemcpyHostToDevice);
}

template <class T, class StringType>
void cudaMallocAndMemcpyData(T**& pointers, const std::vector<std::vector<T>>& vectors, StringType&& string)
{
    for (uint8_t currentPointerNumber{}; currentPointerNumber < vectors.size(); currentPointerNumber++)
    {
        auto cudaStatus = cudaMallocAndMemcpyData(pointers[currentPointerNumber], vectors.at(currentPointerNumber),
            "data"s + std::to_string(currentPointerNumber));
        if (cudaStatus != hipSuccess)
        {
            throw std::runtime_error("hipMemcpy for "s + std::forward<StringType>(string)
                + std::to_string(currentPointerNumber) + " failed!\n" + hipGetErrorString(cudaStatus));
        }
    }
}

template <typename T, typename StringType>
std::vector<T> loadingFileInHEX(StringType&& nameFile)
{
    std::ifstream in(nameFile);
    auto name = "File "s + std::forward<StringType>(nameFile);
    std::vector<T> vector;
    if (!in.fail())
    {
        T buffer{};
        while (in >> std::hex >> std::uppercase >> buffer)
        {
            vector.emplace_back(buffer);
        }
        in.close();
    }
    else
    {
        in.close();
        throw std::runtime_error(name + " is not found!");
    }
    if (vector.size() == 0)
    {
        throw std::runtime_error(name + " is empty or contains wrong data!");
    }

    return vector;
}

template <typename StringType>
std::vector<uint8_t> loadingBytesInHEXFromFile(StringType&& nameFile)
{
    std::ifstream in(nameFile);
    auto name = "File "s + std::forward<StringType>(nameFile);
    std::vector<uint8_t> byteVector{};
    if (!in.fail())
    {
        HEXByteSymbol byte{};
        while (in >> byte)
        {
            byteVector.emplace_back(static_cast<uint8_t>(byte));
        }
        in.close();
    }
    else
    {
        in.close();
        throw std::runtime_error(name + " is not found!");
    }
    if (byteVector.size() == 0)
    {
        throw std::runtime_error(name + " is empty or contains wrong data!");
    }
    return byteVector;
}

template <typename StringType>
void addEntryIntoFile(CRC16&& data, StringType&& nameFile)
{
    std::ofstream output(nameFile, std::ios_base::app);
    if (!output)
    {
        output.close();
        throw std::runtime_error("Writing into file " + std::forward<StringType>(nameFile) + " is impossible!");
    }
    output << std::move(data) << '\n';
    output.close();
}

CRC16 bruteForceCRC16WithGPU(const uint16_t finalXORValue, const std::vector<std::vector<uint8_t>>& data,
    const std::vector<std::vector<uint8_t>>& reflectedData, const std::vector<uint16_t>& crcs)
{
    hipError_t cudaStatus{};
    uint16_t* crcsPointer{};
    CRC16* result{};
    auto sizes = new size_t[4u];
    auto dataPointers = new uint8_t*[4u];
    auto reflectedDataPointers = new uint8_t*[4u];
    try
    {
        cudaStatus = hipSetDevice(0);
        if (hipSetDevice(0) != hipSuccess) {
            throw std::runtime_error("hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n"s
                + hipGetErrorString(cudaStatus));
        }

        for (uint8_t currentSizeNumber{}; currentSizeNumber < data.size(); currentSizeNumber++)
        {
            sizes[currentSizeNumber] = data.at(currentSizeNumber).size();
        }

        cudaMallocAndMemcpyData(dataPointers, data, "data"s);
        cudaMallocAndMemcpyData(reflectedDataPointers, reflectedData, "reflected data"s);

        cudaStatus = cudaMallocAndMemcpyData(crcsPointer, crcs, "crcs"s);
        if (cudaStatus != hipSuccess)
        {
            throw std::runtime_error("hipMemcpy for crcs failed!\n"s + hipGetErrorString(cudaStatus));
        }

        CRC16 defaultCRC16Value{};
        cudaStatus = cudaMallocAndMemcpyData(result, defaultCRC16Value, "result"s);
        if (cudaStatus != hipSuccess)
        {
            throw std::runtime_error("hipMemcpy for result failed!\n"s + hipGetErrorString(cudaStatus));
        }

        uint16_t polynomeCount{ 0xFFFFu };
        uint16_t initValueCount{ 0xFFFFu };
        uint8_t inputOrResultReflectedCombinationCount{ 4u };
        dim3 threadPerBlock(32u, 16u, 1u);
        dim3 blocks(polynomeCount / threadPerBlock.x, initValueCount / threadPerBlock.y,
            inputOrResultReflectedCombinationCount / threadPerBlock.z);

        findCRC16Parameters<<<blocks, threadPerBlock>>>(dataPointers[0], dataPointers[1], dataPointers[2],
            dataPointers[3], reflectedDataPointers[0], reflectedDataPointers[1], reflectedDataPointers[2],
            reflectedDataPointers[3], crcsPointer, sizes[0], sizes[1], sizes[2], sizes[3], finalXORValue, result);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            throw std::runtime_error("Kernel computing failed!\n"s + hipGetErrorString(cudaStatus));
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
        {
            throw std::runtime_error("hipDeviceSynchronize failed!\n"s + hipGetErrorString(cudaStatus));
        }

        CRC16 computedResult{};
        cudaStatus = hipMemcpy(&computedResult, result, sizeof(CRC16), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            throw std::runtime_error("hipMemcpy for computed result failed!\n"s + hipGetErrorString(cudaStatus));
        }

        hipFree(result);
        hipFree(crcsPointer);
        for (uint8_t currentPointerNumber{}; currentPointerNumber < data.size(); currentPointerNumber++)
        {
            hipFree(dataPointers[currentPointerNumber]);
            hipFree(reflectedDataPointers[currentPointerNumber]);
        }
        delete[] sizes;
        delete[] dataPointers;
        delete[] reflectedDataPointers;

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            throw cudaDeviceResetException("hipDeviceReset failed!\n"s + hipGetErrorString(cudaStatus));
        }

        return computedResult;
    }
    catch (std::runtime_error ex)
    {
        hipFree(result);
        hipFree(crcsPointer);
        for (uint8_t currentPointerNumber{}; currentPointerNumber < data.size(); currentPointerNumber++)
        {
            hipFree(dataPointers[currentPointerNumber]);
            hipFree(reflectedDataPointers[currentPointerNumber]);
        }
        delete[] sizes;
        delete[] dataPointers;
        delete[] reflectedDataPointers;

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess)
        {
            std::cerr << "hipDeviceReset failed! " << hipGetErrorString(cudaStatus) << '\n';
        }
        throw;
    }
}

bool XNOR(const bool firstCondition, const bool secondCondition)
{
    return firstCondition == secondCondition;
}

template <typename T>
std::vector<T> reflect(const std::vector<T>& values)
{
    std::vector<T> reflected;
    auto bitCountOfType = sizeof(T) * 8;
    for (const auto& value : values)
    {
        T reflectedValue{};
        for (size_t i{}; i < bitCountOfType; i++)
        {
            uint8_t bitValue = value & (1 << i) ? 1 : 0;
            reflectedValue |= bitValue << ((bitCountOfType - 1) - i);
        }
        reflected.emplace_back(reflectedValue);
    }
    return reflected;
}

void calculateCRC16WithGPU(std::vector<std::vector<uint8_t>>&& data, std::vector<uint16_t>&& crcs)
{
    std::vector<std::vector<uint8_t>> reflectedData
    {
        reflect(data.at(0)),
        reflect(data.at(1)),
        reflect(data.at(2)),
        reflect(data.at(3))
    };
    bool overflowed{};
    uint16_t finalXORValue{ 0xFFFFu };
    auto resultNameFile = "Results.txt"s;
    try {
        auto crcResults = loadingFileInHEX<CRC16>(resultNameFile);
        if (!crcResults.empty())
        {
            const auto& theLastResult = crcResults.back();
            overflowed = true;
            finalXORValue = theLastResult.getFinalXORValue();
        }
    }
    catch (std::runtime_error ex)
    {}
    for (; XNOR(finalXORValue < 0xFFFFu, overflowed); finalXORValue++)
    {
        auto result = bruteForceCRC16WithGPU(finalXORValue, data, reflectedData, crcs);
        if (result.isInitialized())
        {
            std::cout << '\n' << result << '\n';
            addEntryIntoFile(std::move(result), std::move(resultNameFile));
        }
        auto percent = overflowed ? std::trunc(10000 * (static_cast<float>(finalXORValue) / 0xFFFFu)) / 100 : 0;
        std::cout << '\r' << "Completed: " << std::dec << std::setw(6) << percent << "% Final XOR value: "
            << std::noshowbase << std::hex << std::uppercase << finalXORValue;
        if (finalXORValue == 0xFFFFu)
        {
            overflowed = true;
        }
    }
}

void initAndStartCalculating()
{
    std::vector<std::vector<uint8_t>> data
    {
        loadingBytesInHEXFromFile("1.txt"s),
        loadingBytesInHEXFromFile("2.txt"s),
        loadingBytesInHEXFromFile("3.txt"s),
        loadingBytesInHEXFromFile("4.txt"s)
    };
    auto crcs = loadingFileInHEX<uint16_t>("CRC.txt"s);
    calculateCRC16WithGPU(std::move(data), std::move(crcs));
}

int main()
{
    try
    {
        std::cout << "CRC16 brute force with GPU...\n";
        initAndStartCalculating();
    }
    catch (std::runtime_error ex)
    {
        std::cerr << "Error! " << ex.what() << '\n';
        return -1;
    }
    catch (cudaDeviceResetException ex)
    {
        std::cerr << "CUDA device reset error! " << ex.what() << '\n';
        return 1;
    }
    catch (std::exception ex)
    {
        std::cerr << "Unknown error! " << ex.what() << '\n';
        return -1;
    }
    return 0;
}